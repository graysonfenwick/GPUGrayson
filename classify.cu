#include "hip/hip_runtime.h"
/*  
In addition to implementing the device code, answer the following questions:

1) The current value for BINS in config.h is 16.  If you reduce the value for BINS to
8 or 4, how is this going to impact the speedups?  Why?  (You can try this
out if you like.  See models.h for directions.)


2) If more models are added to the program, how is this going to impact 
the speedups?  Why?

3) In regards to the classification, what is the advantage of having fewer
bins then the image pixel space (256 * 256 * 256)?


4) What can go wrong if there are too few bins?

*/

#include <sys/stat.h>
#include <stdlib.h>
#include <stdio.h>
//config.h contains a number of needed definitions
#include "config.h"  
#include "histogram.h"
#include "wrappers.h"
#include "h_classify.h"
#include "d_classify.h"

#include "models.h"

//prototypes for functions in this file 
static void parseCommandArgs(int, char **, char **, int *, char *);
static void printUsage();
static void readPPMImage(char *, unsigned char **, int *, int *, int *, int, int *);
static void writeHistogram(histogramT *, char *);
static void writeBin(FILE *, int *, int);
static void buildName(const char * , char name[NAMELEN]);
static void compareHistograms(histogramT *, histogramT *, int);
static void compareComparisons(comparisonT *, comparisonT *, int);
static void initHistogram(char *, histogramT *, int);
static void printTopTwo(comparisonT *);
static void printTitle(const char *);

/*
    main 
    Opens the ppm file and reads the contents.  Uses the CPU
    to build a histogram of the image, optionally outputting
    the histogram to a file in the form of a C struct initialization.  
    If the save option is not provided the program will also classify
    the image on the CPU and histogram and classify the image on the GPU.
    It compares the CPU and GPU results to make sure they match
    and outputs the times it takes on the CPU and the GPU to build the 
    histogram and perform the classification.
*/
int main(int argc, char * argv[])
{
    unsigned char * hPin, *dPin; 
    histogramT * h_hgram, * d_hgram;
    char * inputfile;
    char outputfile[NAMELEN];
    int width, height, color, pitch, saveOutput;
    float cpuTime, gpuTime;
    int gpuStride = 32, cpuStride = 4;

    printTitle(argv[0]);

    //need an array of these; one for each model
    //one array for the GPU and one array for the CPU
    comparisonT * hresult = (comparisonT *) Malloc(sizeof(comparisonT) * MODELS);
    comparisonT * dresult = (comparisonT *) Malloc(sizeof(comparisonT) * MODELS);

    parseCommandArgs(argc, argv, &inputfile, &saveOutput, outputfile);

    //create histogram structs for the host and the device
    h_hgram = (histogramT *) Malloc(sizeof(histogramT));
    d_hgram = (histogramT *) Malloc(sizeof(histogramT));
    initHistogram(inputfile, h_hgram, TOTALBINS);


    //read and pitch the image for the CPU
    readPPMImage(inputfile, &hPin, &width, &height, &color, cpuStride, &pitch);

    if (saveOutput) 
    {
       //if save is requested, just save results of histogram to file
       //and don't continue
       printf("\nComputing histogram of %s.\n", inputfile);
       histoOnCPU(h_hgram, hPin, height, width, pitch);
       writeHistogram(h_hgram, outputfile);
       printf("Storing result in %s.\n", outputfile);
       return EXIT_SUCCESS;
    }

    printf("\nComputing histogram and classifying %s.\n", inputfile);

    //use the CPU to build the histogram and classify it
    cpuTime = h_classify(h_hgram, hresult, models, MODELS, hPin, height, width, pitch); 
    printf("\tCPU time: \t\t%f msec\n", cpuTime);

    //read and pitch the image for the GPU
    readPPMImage(inputfile, &dPin, &width, &height, &color, gpuStride, &pitch);

    //use the GPU to build the histogram and classify it
    initHistogram(inputfile, d_hgram, TOTALBINS);
    gpuTime = d_classify(d_hgram, dresult, models, MODELS, dPin, height, width, pitch);
    compareHistograms(d_hgram, h_hgram, TOTALBINS);
    compareComparisons(dresult, hresult, MODELS);
    printf("\tGPU time: \t\t%f msec\n", gpuTime);
    printf("\tSpeedup: \t\t%f\n", cpuTime/gpuTime);
    printTopTwo(dresult);

    free(d_hgram);
    free(h_hgram);
    free(hPin);
    free(dPin);
    return EXIT_SUCCESS;
}

void printTitle(const char * executable)
{
    /*
   if (strcmp(executable, "./classify1") == 0)
      printf("Classify using a naive sum to calculate the histogram size\n");
   else if (strcmp(executable, "./classify2") == 0)
      printf("Classify using a Kogge-Stone sum to calculate the histogram size\n");
   else if (strcmp(executable, "./classify3") == 0)
      printf("Classify using a Belloch sum to calculate the histogram size\n");
   else if (strcmp(executable, "./classify4") == 0)
      printf("Classify using a Belloch sum with reduced bank conflicts to calculate the histogram size\n");
    */
}
   




/*
    initHistogram
    Initializes a histogram struct by setting the bin values to 0 and setting
    the fileName field to the name of the file containing the image to histogram.
*/
void initHistogram(char * fileName, histogramT * histP, int length)
{
    int i;
    strncpy(histP->fileName, fileName, sizeof(histP->fileName));
    for (i = 0; i < length; i++)
    {
       histP->histogram[i] = 0;
    }
}

/*
    printTopTwo
    Finds and prints the top two matches in the comparison struct.
    comparison values will range from 0 to 1.
    An exact match will have a comparison value of 1.0, which indicates
    the model matches the input image exactly.  
    A comparison value of .8 means that %80 of the pixels in the
    model and the input image are the same.
*/
   
void printTopTwo(comparisonT * result)
{
    int first = -1, second = -1;
    int i;
    for (i = 0; i < MODELS; i++)
    {
        if (first == -1)  //both first and second are -1
            first = i;
        else if (second == -1) //first is not -1
        {
            if (result[i].comparison > result[first].comparison)
            {
                second = first;
                first = i;
            } else
            {
                second = i;
            }
        } else if (result[i].comparison > result[first].comparison)
        {
            second = first;
            first = i;
        } else if (result[i].comparison > result[second].comparison)
        {
            second = i;
        }
    }
    printf("\nMatches\n");
    printf("-------\n");
    printf("\tFirst:  %s    \t%5.1f%%\n", result[first].fileName, 
           (result[first].comparison * 100));
    printf("\tSecond: %s    \t%5.1f%%\n", result[second].fileName, 
           (result[second].comparison * 100));
}

/* 
    compareHistograms
    This function takes two histogramT structs. One histogramT 
    contains bins calculated  by the GPU. The other histogramT
    contains bins calculated by the CPU. This function examines
    each bin to see that they match.

    d_Pout - histogram calculated by GPU
    h_Pout - histogram calculated by CPU
    length - number of bins in histogram
    
    Outputs an error message and exits program if the histograms differ.
*/
void compareHistograms(histogramT * d_Pout, histogramT * h_Pout, int length)
{
    int i;
    for (i = 0; i < length; i++)
    {
        if (d_Pout->histogram[i] != h_Pout->histogram[i])
        {
            printf("Histograms don't match.\n");
            printf("host bin[%d] = %d\n", i, h_Pout->histogram[i]);
            printf("device bin[%d] = %d\n", i, d_Pout->histogram[i]);
            exit(EXIT_FAILURE);
        }
    }
}

/* 
    compareComparisons
    This function takes two comparisonT structs. One comparisonT 
    contains a comparison array calculated  by the GPU.  The other 
    comparsionT contains a comparison array calculated
    by the CPU.  This function examines each comparison array
    element to see that they match.

    d_Pout - comparison calculated by GPU
    h_Pout - comparison calculated by CPU
    length - number of comparison
    
    Outputs an error message and exits program if the comparisons differ.
*/
void compareComparisons(comparisonT * d_Pout, comparisonT * h_Pout, int length)
{
    int i;
    for (i = 0; i < length; i++)
    {
        if (abs(d_Pout[i].comparison - h_Pout[i].comparison) > 0.01)
        {
            printf("Comparisons don't match for %s.\n", d_Pout[i].fileName);
            printf("host comparison[%d] = %f\n", i, h_Pout[i].comparison);
            printf("device comparison[%d] = %f\n", i, d_Pout[i].comparison);
            exit(EXIT_FAILURE);
        }
    }
}

/* 
    writeHistogram
    Writes a histogram to an output file.

*/
void writeHistogram(histogramT * histP, char * outfile)
{
    FILE *fp = fopen(outfile, "w");
    if (fp == NULL)
    {
        printf("\nUnable to open output file: %s\n", outfile);
        printUsage();
    }
    char varname[NAMELEN];
    buildName(histP->fileName, varname);
    fprintf(fp, "histogramT %s =\n{\n\"%s.ppm\",\n", varname, varname);
    writeBin(fp, histP->histogram, TOTALBINS);
    fprintf(fp, "\n};");
    fclose(fp);
}

/*
   buildName
   Used to strip off the final four characters (.ppm) in a file
   name and any leading characters up to and including the
   last / to build a name that is then stored in the output file
   with the histogram.

   example: buildName("images/CaptainAmerica1.ppm", varname)
            stores "CaptainAmerica1" in varname
*/ 
void buildName(const char * outfile, char varname[NAMELEN])
{
    int endPoint = strlen(outfile) - 4;
    int startPoint = endPoint;
    //decrement startPoint until it reaches beginning of string
    //or a /
    while (startPoint > 0)
    {
        if (outfile[startPoint] == '/') {startPoint++; break;}
        startPoint--;
    }
    strcpy(varname, &outfile[startPoint]);
    varname[strlen(varname)-4] = '\0';
    
}

/*
   writeBin
   Outputs the bin to the output file.
*/
void writeBin(FILE * fp, int * bin, int length)
{
   int i;
   fprintf(fp, "{\n");
   for (i = 0; i < length - 1; i++)
   {
        fprintf(fp, "%d, ", bin[i]);
        if ((i + 1) % 32 == 0) fprintf(fp, " /* %d-%d */\n", i - 31, i);
   }
   fprintf(fp, "%d /* %d-%d */\n}", bin[i], i - 31, i);
}

/*
    readPPMImage
    This function opens a ppm file and reads the contents.  A ppm file
    is of the following format:
    P6
    width  height
    color
    pixels

    Each pixel consists of bytes for red, green, and blue.  If color
    is less than 256 then each color is encoded in 1 byte.  Otherwise,
    each color is encoded in 2 bytes. This function fails if the color
    is encoded in 2 bytes.
    
    The array Pin is initialized to the pixel bytes.  width, height,
    and color are pointers to ints that are set to those values.
    filename - name of the .ppm file

    If stride is not 1 then the array to hold the pixels is pitched
    so that the pitch is greater than or equal to width and also a 
    multiple of the stride.  The stride represents the memory burst length.
*/
void readPPMImage(char * filename, unsigned char ** Pin, 
                  int * width, int * height, int * color, 
                  int stride, int * pitch)
{
    int ht, wd, ptch, colr;
    char P6[3];
    FILE * fp = fopen(filename, "rb"); //read binary
    int count = fscanf(fp, "%s\n%d %d\n%d\n", P6, &wd, &ht, &colr);

    //should have read four values
    //first value is the string "P6"
    //color value must be less than 256 and greater than 0
    if (count != 4 || strncmp(P6, "P6", CHANNELS) || colr <= 0 || colr > 255)
    {
        printf("\nInvalid file format.\n\n");
        printUsage();
    }

    //pitch is a multiple of the stride
    ptch = ceil(wd/(float)stride) * stride;
       
    (*Pin) = (unsigned char *) Malloc(sizeof(unsigned char) * ptch * ht * CHANNELS);
    for (int i = 0; i < ht; i++)
    {
        if (fread(&(*Pin)[i * ptch * CHANNELS], 
                  sizeof(unsigned char) * wd * CHANNELS, 1, fp) != 1)
        {
            printf("Invalid file format.\n\n");
            printUsage();
        }
    }

    (*width) = wd;
    (*height) = ht;
    (*color) = colr;
    (*pitch) = ptch;
    fclose(fp);
}

/*
    parseCommandArgs
    This function parses the command line arguments. The program can be executed 
    like this:
    ./classify [-s <outfile>]  <file>.ppm
    or
    ./classify <file>.ppm
    If the -s option is provided, the histogram is simply built using the CPU
    and the result is stored in the output file. No classification is performed.
    In addition, it checks to see if the last command line argument
    is a ppm file and sets (*fileNm) to argv[i] where argv[i] is the name of the ppm
    file.  
*/
void parseCommandArgs(int argc, char * argv[], char ** fileNm, int * saveOutput,
                      char outputFile[NAMELEN])
{
    int fileIdx = argc - 1, save = 0;
    struct stat buffer;

    for (int i = 1; i < argc - 1; i++)
    {
        
        if (strncmp("-s", argv[i], 3) == 0) 
        {
            save = 1;
            if (i+1 >= argc - 1) 
            {
                printf("Invalid output file name: %s.\n", argv[i+1]);
                printUsage();
            }
            strncpy(outputFile, argv[i+1], NAMELEN);
            i++;
        } else if (strncmp("-h", argv[i], 3) == 0) 
        {
            printUsage();
        } else  
            printUsage();
    } 

    //check the input file name (must end with .ppm)
    int len = strlen(argv[fileIdx]);
    if (len < 5) printUsage();
    if (strncmp(".ppm", &argv[fileIdx][len - 4], 4) != 0) printUsage();

    //stat function returns 1 if file does not exist
    if (stat(argv[fileIdx], &buffer)) printUsage();
    (*fileNm) = argv[fileIdx];
    (*saveOutput) = save;
}

/*
    printUsage
    This function is called if there is an error in the command line
    arguments or if the .ppm file that is provided by the command line
    argument is improperly formatted.  It prints usage information and
    exits.
*/
void printUsage()
{
    printf("This application takes as input the name of a .ppm\n");
    printf("file containing a color image and creates a histogram\n");
    printf("of the image. It then computes an intersection of this histogram\n");
    printf("and the other histograms defined in 'models.h'. It outputs the\n");
    printf("names of the two best matching images. This work is\n");
    printf("performed on the CPU and the GPU. Their results are timed and\n");
    printf("compared.\n");
    printf("\nusage: ./classify [-s <outfile>] <name>.ppm\n");
    printf("       If the -s argument is provided, the histogram is saved\n");
    printf("              in the output file and no classification is performed.\n");
    printf("              This is used to build model histograms for this program.\n");
    printf("       <name>.ppm is the name of the input ppm file.\n");
    printf("Examples:\n");
    printf("./classify images/WonderWoman1.ppm\n");
    printf("./classify -s WonderWoman1.h images/WonderWoman1.ppm\n");
    exit(EXIT_FAILURE);
}

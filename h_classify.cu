#include "hip/hip_runtime.h"
#include <stdio.h>
#include "config.h"
#include "histogram.h"
#include "h_classify.h"
#include "CHECK.h"
#include "wrappers.h"

static void classifyOnCPU(histogramT **, int, histogramT *, comparisonT *); 
static void normalizeHistogram(float *, histogramT *, int);
static float computeIntersection(float *, float *, int);
static int sumInts(int *, int);
static void printFloatArray(float * array, int startIdx, int length);

/*
   h_classify
   Builds a histogram of an input image and classifies the
   histogram using the provided models.

   Outputs:
   Phisto - pointer to a histogram structure that will be set
            to the histogram built
   results - an array of structs where each struct contains the name of
             the model used to compute the comparison and the comparison result

   Inputs:
   models - an array of histograms that is used to perform the classification
   modelCt - count of the number of models
   Pin - array that contains the color pixels.
   width and height - dimensions of the image.
   pitch - length of each row of the image (may be larger than width)
*/
float h_classify(histogramT * Phisto, comparisonT * results,
                 histogramT ** models, int modelCt, unsigned char * Pin, 
                 int height, int width, int pitch) 
{
    hipEvent_t start_cpu, stop_cpu;
    float cpuMsecTime = -1;

    //Use cuda functions to do the timing 
    //create event objects
    CHECK(hipEventCreate(&start_cpu));
    CHECK(hipEventCreate(&stop_cpu));

    //record the starting time
    CHECK(hipEventRecord(start_cpu));

    //first calculate histogram
    histoOnCPU(Phisto, Pin, height, width, pitch);

    //now, classify it
    classifyOnCPU(models, modelCt, Phisto, results);
 
    //record the ending time and wait for event to complete
    CHECK(hipEventRecord(stop_cpu));
    CHECK(hipEventSynchronize(stop_cpu));
    //calculate the elapsed time between the two events 
    CHECK(hipEventElapsedTime(&cpuMsecTime, start_cpu, stop_cpu));
    return cpuMsecTime;
}

/*
   histoOnCPU
   Performs the histo of an image on the CPU.

   Output:
   Phisto - pointer to a histogram structure that will be set
            to the histogram built
   Inputs:
   Pin - array that contains the color pixels.
   width and height - dimensions of the image.
   pitch - length of each row of the image (may be larger than width)
*/
void histoOnCPU(histogramT * Phisto, unsigned char * Pin, int height, 
               int width, int pitch)
{
    unsigned char redVal, greenVal, blueVal;
    int j, i; 

    //calculate the row width of the input 
    int rowWidth = CHANNELS * pitch;
    for (j = 0; j < height; j++)
    {
        for (i = 0; i < width; i++)
        {
            //use red, green, and blue values to compute bin number
            redVal = Pin[j * rowWidth + i * CHANNELS]; 
            greenVal = Pin[j * rowWidth + i * CHANNELS + 1]; 
            blueVal = Pin[j * rowWidth + i * CHANNELS + 2]; 
            int bin = (redVal/TONESPB)*BINS*BINS + (blueVal/TONESPB)*BINS
                      + greenVal/TONESPB;
            Phisto->histogram[bin]++; 
        }
    }
}

/* 
    classifyOnCPU   
    Takes as input a histogram and array of model histograms and compares the input
    histogram to each model by calculating an intersection. The result of each
    comparison and the name of the model is stored in the results array.

    Inputs:
    models - array of histograms to use for the comparison
    input - input histogram to be compared to the others
    Outputs:
    results - result of the comparisons
*/ 
void classifyOnCPU(histogramT ** models, int modelCt, histogramT * input, 
                   comparisonT * results)
{
    int i = 0;

    float intersection;
    float * normInput = (float *) Malloc(sizeof(float) * TOTALBINS);
    float * normModel = (float *) Malloc(sizeof(float) * TOTALBINS);

    //since images may be different sizes, their histograms need to be normalized
    //to a common size
    //first, normalize the input histogram
    normalizeHistogram(normInput, input, TOTALBINS);
    for (i = 0; i < modelCt; i++)
    {
        //normalize the model used in the comparison
        normalizeHistogram(normModel, models[i], TOTALBINS); 

        //compare normalized input to normalized model
        intersection = computeIntersection(normInput, normModel, TOTALBINS);

        //calculate and store the result of the comparison
        results[i].comparison = intersection/NORMMAX;
        strcpy(results[i].fileName, models[i]->fileName);
    }
}

/*
    computeIntersection
    This function returns the intersection of the two histograms. 
    If a pixel is in an intersection then the pixel appears in both the
    images.

    Inputs:
    model - histogram to use for intersection
    input - histogram to use for intersection
    numBins - number of bins in each histogram array
    Outputs:
    returns intersection
*/
float computeIntersection(float * model, float * input, int numBins)
{
    float result = 0;
    int i;
    for (i = 0; i < numBins; i++)
    {
        //For example, RGB i appears Y times in input and Z times in model
        //and therefore at least min(Y,Z) times in each
        result += min(input[i], model[i]);
    }
    return result; 
}

/*
   sumInts
   Returns the sum of an array of ints.

   Inputs:
   array - pointer to an array of ints
   length - length of an array
   Output:
   sum of array
*/
int sumInts(int * array, int length)
{
    int sum = 0;
    int i;
    for (i = 0; i < length; i++) sum += array[i];
    return sum;
}

/*
    normalizeHistograms
    This function produces an output array of floats that is the normalization
    of the data in the input array. If the range of input bin values
    is from 0 to MAX then the bin values in the output array will range from
    0 to NORMMAX. 
    Inputs:
    input - pointer to a struct containing the histogram bins 
    length - number of bins in the histogram
    Output:
    normData - normalization of the output data
*/
void normalizeHistogram(float * normData, histogramT * input, int length)
{
    //get the count of the number of pixels in the histogram
    //by adding up all of the bins
    int pixels = sumInts(input->histogram, TOTALBINS);
    int i;
    for (i = 0; i < length; i++)
    {
        normData[i] = (input->histogram[i]/(float)pixels) * NORMMAX;
    }
}

//can be used for debugging
void printFloatArray(float * array, int startIdx, int length)
{
    int i, j = 0;
    for (i = startIdx; i < startIdx + length; i++, j++)
    {
        if ((j % 16) == 0) printf("\n%3d: ", i);
        printf("%6.1f ", array[i]);
    }
    printf("\n");
}


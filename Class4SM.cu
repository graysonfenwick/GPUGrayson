#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "config.h"
#include "histogram.h"
#include "d_classify.h"
#include "CHECK.h"
#include "wrappers.h"

//Our CUDA machine has compute capability 6.1
//These are the number of memory banks of a machine with that capability
//according to the Nvidia documentation
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#define CLASSBLOCKDIM 512
//shared memory size needs to be twice CLASSBLOCKDIM + one integer for
//every 32 integers
#define SHAREDMEMSIZE ((CLASSBLOCKDIM << 1) + ((CLASSBLOCKDIM << 1) >> LOG_NUM_BANKS))  
static __device__ void bankBelloch(float *histogram, int *histSz);

//parameters for building the histogram from the image
//TILEWIDTH is number of pixels in a row that a single thread will handle
#define TILEWIDTH 8 
#define HISTBLOCKDIM 32

//prototypes for functions local to this file
static float histogramOnGPU(histogramT *, unsigned char *, int, int, int);
static float classifyOnGPU(float *, int *, int modelCt);
static void normalizeHost(float *, int *);
//prototypes for the kernels
static __global__ void d_histoKernel(histogramT *, unsigned char *, int, int, int);
static __global__ void d_classifyKernel(float *, float *, int *);
static __global__ void emptyKernel();

//prototypes of functions called by d_classifyKernel

static __device__ void normalizeHist(float *, int);
static __device__ void intersection(float * normHistograms, float * intersect);

//for debugging
static __device__ void printFloatArray(float * array, int startIdx, int length);
__device__ void printIntArray(int * data, int length, const char * label);

/*
    d_classify
    Performs image classification on the GPU by first building a histogram
    to represent the image and then comparing the histogram to each of the
    histogram models.

    Outputs:
    Phisto - pointer to histogramT struct containing the bins 
    dresult - comparisonT array of structs; one element per model

    Inputs:
    models - an array of pointers to histogramT structs; one element per
             model to be compared to the input
    Pin - array contains the color pixels of the image to be used for 
          building a histogram and doing the classification
    width and height - dimensions of the image
    pitch - size of each row
 
    Returns the amount of time it takes to build the histogram and
      classify the image
*/
float d_classify(histogramT * Phisto, comparisonT * dresult, 
                 histogramT ** models, int modelCt, unsigned char * Pin,
                 int height, int width, int pitch) 
{
    float gpuMsecTime1, gpuMsecTime2;

    //launch an empty kernel to get more accurate timing
    emptyKernel<<<1024, 1024>>>();

    //build a histogram of the input image
    gpuMsecTime1 = histogramOnGPU(Phisto, Pin, height, width, pitch);

    //allocate array to hold all histograms, including the histogram for the input
    int * histograms = (int *) Malloc(sizeof(int) * (modelCt + 1) * TOTALBINS);

    //copy the histogram for the input to the beginning of the array
    memcpy(histograms, Phisto->histogram, sizeof(int) * TOTALBINS);

    //copy the remaining histograms
    for (int i = 1; i <= modelCt; i++) 
        memcpy(&histograms[i*TOTALBINS], models[i - 1]->histogram, sizeof(int) * TOTALBINS);

    //allocate an array of floats to hold the comparisons
    float * comparisons = (float *) Malloc(sizeof(int) * modelCt);

    //perform the classification
    gpuMsecTime2 = classifyOnGPU(comparisons, histograms, modelCt);

    //copy the results into the output
    for (int i = 0; i < modelCt; i++)
    {
        dresult[i].comparison = comparisons[i];
        strncpy(dresult[i].fileName, models[i]->fileName, NAMELEN);
    }

    return gpuMsecTime1 + gpuMsecTime2;
}

/*
   histogramOnGPU
   Builds a histogram to represent the input image.

   Outputs:
   Phisto - pointer to the histogramT struct containing the bins

   Inputs:
   Pin - array contains the color pixels of the image to be used for 
         building a histogram
   width and height -  dimensions of the image
   pitch - size of each row
 
   Returns the amount of time it takes to build the histogram 
*/
float histogramOnGPU(histogramT * Phisto, unsigned char * Pin, int height, 
                     int width, int pitch)
{
    //THIS CODE IS COMPLETE

    hipEvent_t start_gpu, stop_gpu;
    float gpuMsecTime = -1;
    
    //Use cuda functions to do the timing 
    //create event objects
    CHECK(hipEventCreate(&start_gpu));
    CHECK(hipEventCreate(&stop_gpu));

    unsigned char * d_Pin;
    int numPinBytes = sizeof(unsigned char) * pitch * height * CHANNELS;
    histogramT * d_Phisto;
    
    //create the array on the GPU to hold input
    CHECK(hipMalloc((void **)&d_Pin, numPinBytes));
    CHECK(hipMemcpy(d_Pin, Pin, numPinBytes, hipMemcpyHostToDevice));

    //create the array on the GPU to hold the histogram
    CHECK(hipMalloc((void **)&d_Phisto, sizeof(histogramT)));
    CHECK(hipMemcpy(d_Phisto, Phisto, sizeof(histogramT),
          hipMemcpyHostToDevice));
    
    //build the histogram
    CHECK(hipEventRecord(start_gpu));

    //each thread calculates TILEWIDTH elements in a row
    dim3 grid(ceil(width/(float)(HISTBLOCKDIM * TILEWIDTH)),
              ceil(height/(float)HISTBLOCKDIM), 1);
    dim3 block(HISTBLOCKDIM, HISTBLOCKDIM, 1);

    d_histoKernel<<<grid, block>>>(d_Phisto, d_Pin, height, width, pitch);

    CHECK(hipEventRecord(stop_gpu));
    CHECK(hipMemcpy(Phisto, d_Phisto, sizeof(histogramT),
          hipMemcpyDeviceToHost));
    //record the ending time and wait for event to complete
    CHECK(hipEventSynchronize(stop_gpu));
    //calculate the elapsed time between the two events 
    CHECK(hipEventElapsedTime(&gpuMsecTime, start_gpu, stop_gpu));

    return gpuMsecTime;
}

/*
   d_histoKernel
   Kernel code executed by each thread on its own data when the kernel is
   launched. Each thread operates on TILEWIDTH pixels in a row.

   Inputs:
   Pin - array contains the color pixels to be used to build the histogram
   width and height - dimensions of the image
   pitch - size of each row

   Output:
   histo - pointer to a histogramT struct that contains an array of bins
*/
__global__
void d_histoKernel(histogramT * histo, unsigned char * Pin, int height,
                  int width, int pitch)
{
    //THIS CODE IS COMPLETE.  You can replace it with a faster version
    //if you like, but the shared memory version won't work with all
    //TOTALBINS sizes.  If you use that one, the largest BIN value can
    //only be 8.

    int colStart = (blockIdx.x * blockDim.x + threadIdx.x) * TILEWIDTH;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col;

    //use a privatization technique to reduce the number of atomic adds
    int accumulator = 0;
    int prevBin = -1;
    int currBin;

    //go through each pixel in the tile
    for (int i = 0; i < TILEWIDTH; i++)
    {
        col = colStart + i;
        if (row < height && col < width)
        {
            //flatten the 2D indices
            int pIndx = row * CHANNELS * pitch + col * CHANNELS;

            unsigned char redVal = Pin[pIndx];
            unsigned char greenVal = Pin[pIndx + 1];
            unsigned char blueVal = Pin[pIndx + 2];
            currBin = (redVal/TONESPB)*BINS*BINS + (blueVal/TONESPB)*BINS
                       + greenVal/TONESPB;
            if (currBin != prevBin)
            {
                if (accumulator > 0) 
                    atomicAdd(&(histo->histogram[prevBin]), accumulator); 
                prevBin = currBin;
                accumulator = 1;
            } else accumulator++;
        }
    }
    if (accumulator > 0)
    {
        atomicAdd(&(histo->histogram[prevBin]), accumulator); 
    }
}

/*
    classifyOnGPU
    Performs image classification on the GPU

    Outputs:
    comparisons - an array of size modelCt. comparisons[i] is set to the
                  result of comparing the input image to model i
                  The size of this array is modelCt.

    Inputs:
    histograms - an array of histograms. 
        The histogram for the input image is in:
        histograms[0] ... histogram[TOTALBINS - 1]
        The histogram for model 0 is in:
        histograms[TOTALBINS] ... histogram[2*TOTALBINS - 1]

        The histogram for the last model is in:
        histograms[modelCt*TOTALBINS] ... histogram[modelCt*TOTALBINS - 1]
        Thus, note that the array contains the input histogram and the
        model histograms and thus is of size (modelCt + 1) * TOTALBINS
   
    modelCt - count of the number of models used for the classification 
 
    Returns the amount of time it takes to classify the image
*/
float classifyOnGPU(float * comparisons, int * histograms, int modelCt)
{
    hipEvent_t start_gpu, stop_gpu;
    float gpuMsecTime = -1;

    //allocate an float array on the GPU to hold the normalized histograms
    //It needs to be big enough to hold the histogram of the input image and
    //and the histograms of all of the models.
    float * dNormHistograms;
    float * normHistograms = (float *) Malloc(sizeof(float) * (modelCt + 1) * TOTALBINS);
    CHECK(hipMalloc((void **)&dNormHistograms, sizeof(float) * TOTALBINS * (modelCt + 1)));

    int * dhistograms;

    normalizeHost(normHistograms, histograms);
    //allocate an int array on the GPU to hold the original histograms
    //It needs to be big enough to hold the histogram of the input image and
    //and the histograms of all of the models.
    CHECK(hipMalloc((void **)&dhistograms, sizeof(int) * TOTALBINS * (modelCt + 1)));

    //copy input histograms into dhistograms
    CHECK(hipMemcpy(dhistograms, histograms, sizeof(int) * TOTALBINS * (modelCt + 1),
          hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dNormHistograms, normHistograms, sizeof(float) * TOTALBINS * (modelCt + 1),
          hipMemcpyHostToDevice));
 
    float * dcomparisons;
    //allocate a float array on the GPU to hold the comparisons
    //there needs to be one element per model
    CHECK(hipMalloc((void **)&dcomparisons, sizeof(float) * modelCt ));
    
    //Use cuda functions to do the timing 
    //create event objects
    CHECK(hipEventCreate(&start_gpu));
    CHECK(hipEventCreate(&stop_gpu));

    //record the starting time
    CHECK(hipEventRecord(start_gpu));

    //each model is handled by a single block of threads
    //an extra block of threads is needed to normalize the input histogram
    dim3 grid(modelCt + 1, 1, 1);
    //don't make block any larger than the number of bins
    dim3 block(min(TOTALBINS, CLASSBLOCKDIM), 1);
    //dim3 block(1024, 1);
    d_classifyKernel<<<grid, block>>>(dcomparisons, dNormHistograms, dhistograms);

    CHECK(hipEventRecord(stop_gpu));

    //copy the device comparison array into the host comparison array
    CHECK(hipMemcpy(comparisons, dcomparisons, sizeof(float) * modelCt,
          hipMemcpyDeviceToHost));

    //record the ending time and wait for event to complete
    CHECK(hipEventSynchronize(stop_gpu));
    //calculate the elapsed time between the two events 
    CHECK(hipEventElapsedTime(&gpuMsecTime, start_gpu, stop_gpu));

    return gpuMsecTime;
}
void normalizeHost(float * normHist, int * hist) {
    int i;
    int sum = 0;
    for (i = 0; i < TOTALBINS; i++) {
        sum += hist[i];
    }
    for (i = 0; i < TOTALBINS; i++) {
        normHist[i] = (hist[i]/(float)sum) * NORMMAX;
    }
}

/*
    d_classifyKernel
    Kernel used to do the image classification on the GPU.  Each block of
    threads normalizes a single histogram. After that, every block except
    for block 0 will perform the intersection and store a
    result in the comparisons array.
    Thus, each block (except for 0) produces one result for the comparisons
    array.  Each thread in a block handles TOTALBINS/blockDim.x elements
 
    Inputs: 
    histograms - array of size gridDim.x * TOTALBINS. It contains
                 gridDim.x histograms each of size TOTALBINS.  The first one 
                 is the input histogram.
    Outputs:
    comparisons - comparison[i] is set to the value of the comparison of the
                  input histogram and the histogram of model i; for example,
                  comparison[0] is set to comparison of the input and model 0.
    normHistograms - array of size gridDim.x * TOTALBINS.  It contains
                     gridDim.x histograms that are equal to the normalization
                     of the input histograms.
*/

__device__ int blockSync = 0;   //need this to provide synchronization among blocks
__global__ void d_classifyKernel(float * comparisons, float * normHistograms, int * histograms) 
{
    __shared__ float sNormHistograms[TOTALBINS * 2];
    __shared__ int histSz;
    __shared__ float intersect;

    //thread 0 in the block should initialize histSz and intersect to 0
    if (threadIdx.x == 0)
    {
       intersect = 0;
       histSz = 0;
    }
    __syncthreads();

    int idx;
    int * x = &histograms[TOTALBINS * (blockIdx.x + 1)];
    for (idx = threadIdx.x; idx < TOTALBINS; idx += blockDim.x) {
        sNormHistograms[idx] = normHistograms[idx];
        sNormHistograms[idx + TOTALBINS] = x[idx];
    }
    
    int prevSz;
    int i = 0;
    int bins = TOTALBINS;
    int factor = 2;   //each thread handles 2 elements in the bankBelloch
    while (bins > 0)
    {  
       bankBelloch(&sNormHistograms[TOTALBINS + i * blockDim.x * factor], 
                   &prevSz);
       __syncthreads();
       if (threadIdx.x == 0) 
       {
          histSz = histSz + prevSz;
       }
       bins -= (blockDim.x * factor);
       i+=1;
       __syncthreads();
    }
    //normalize the histogram 
    normalizeHist(sNormHistograms, histSz);
    __syncthreads();

    intersection(sNormHistograms, &intersect);
    __syncthreads();
    if (threadIdx.x == 0) 
    { 
        comparisons[blockIdx.x] = intersect/NORMMAX;
    }
}



/* 
   intersection
   Calculates the intersection of the input histogram and a model histogram
   after they have been normalized.
   The input histogram is in normHistograms[0] ... normHistograms[TOTALBINS - 1]
   The model histogram is in normHistograms[TOTALBINS * blockIdx.x] ...
   normHistograms[TOTALBINS * blockIdx.x - 1]

Inputs:
normHistograms - array of TOTALBINS * gridDim.x bins (gridDim.x histograms)
intersect - pointer to the shared intersect value

Outputs:
shared intersect variable is incremented by the intersection calculated by the
thread running this code 
 */
__device__ void intersection(float * normHistograms, float * intersect)
{
    float * normHistogramTile = &normHistograms[TOTALBINS];
    int tdx = threadIdx.x;
    while (tdx < TOTALBINS)
    {
        float minTwo = fmin(normHistogramTile[tdx], normHistograms[tdx]);
        atomicAdd(intersect, minTwo);
        tdx += blockDim.x;
    }
}

__device__ void printIntArray(int * data, int length, const char * label)

{
    long i, j = 0;
    printf("%s\n", label);
    for (i = 0; i < length; i++, j++)
    {
        if ((j % 10) == 0) printf("\n%3d: ", i);
        printf("%5d ", data[i + CONFLICT_FREE_OFFSET(i)]);
    } 
    printf("\n");
}        

__device__ void bankBelloch(float *histogram, int *histSz)
{
    //add one integer of padding every 32 elements
    __shared__ int shHistogram[SHAREDMEMSIZE];  // allocated on invocation
    int thid = threadIdx.x;
    int offset = 1;
    long n = CLASSBLOCKDIM * 2; 

    long ai = 2*thid;
    long bi = 2*thid + 1;
    long bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    long bankOffsetB = CONFLICT_FREE_OFFSET(bi); 
    shHistogram[ai + bankOffsetA] = histogram[ai];
    shHistogram[bi + bankOffsetB] = histogram[bi];
    /*
       __syncthreads();
       if (threadIdx.x == 0 && blockIdx.x == 0)
       {
       printIntArray(shHistogram, n, "4) After shared memory load");
       }
       __syncthreads();
       return;
     */

    for (int d = n>>1; d > 0; d >>= 1)     // build sum in place up the tree
    { 
        __syncthreads();
        if (thid < d)
        {
            long ai = offset*(2*thid+1)-1;
            long bi = offset*(2*thid+2)-1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);    
            shHistogram[bi] += shHistogram[ai];
        }

        offset *= 2;
    }

    if (thid==0) { shHistogram[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;}
    /*   
         __syncthreads();
         if (threadIdx.x == 0 && blockIdx.x == 0)
         {
         printIntArray(shHistogram, n, "4) After down sweep");
         }
         __syncthreads();
         return;
     */

    for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();
        if (thid < d)                     
        {
            long ai = offset*(2*thid+1)-1;
            long bi = offset*(2*thid+2)-1;
            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);    

            int t = shHistogram[ai];
            shHistogram[ai] = shHistogram[bi];
            shHistogram[bi] += t; 
        }
    }

    __syncthreads();
    /*
       if (threadIdx.x == 0 && blockIdx.x == 0)
       {
       printIntArray(shHistogram, n, "4) After up sweep");
       }
       __syncthreads();
       return;
     */
    if (threadIdx.x == 0)
    {
        (*histSz) = shHistogram[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] +
            histogram[n - 1];
    }
}

/*
   normalizeHist
   Normalizes the histogram so that every bin value is between 0 and NORMMAX.
   The histogram to be normalized is in elements
   histograms[blockIdx.x * TOTALBINS] ... histograms[(blockIdx.x + 1) * TOTALBINS]
   The result will be stored in normHistograms[blockIdx.x * TOTALBINS] ... 
   normHistograms[(blockIdx.x + 1) * TOTALBINS]

Inputs:
histograms - array that holds the histogram to be normalized
histSz - size of the input histogram (sum of its bins)

Outputs:
normHistograms - array to hold the normalized histogram
 */
__device__ void normalizeHist(float * normHistograms, int histSz)
{
    int tdx = threadIdx.x;
    float * normHistogramTile = &normHistograms[TOTALBINS];
    while (tdx < TOTALBINS)
    {
        normHistogramTile[tdx] = (normHistogramTile[tdx]/(float)histSz) * NORMMAX;
        tdx += blockDim.x;
    }
} 

//this can be used for debugging
__device__ void printFloatArray(float * array, int startIdx, int length)
{
    int i, j = 0;
    for (i = startIdx; i < startIdx + length; i++, j++)
    {
        if ((j % 16) == 0) printf("\n%3d: ", i);
        printf("%6.1f ", array[i]);
    } 
}        

//launched to get more accurate timing
__global__ void emptyKernel()
{
}
